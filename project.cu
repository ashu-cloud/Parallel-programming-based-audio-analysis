
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <string>
#include <cstdlib>
#include <complex>
#include <cmath>
#include <vector>
#include <valarray>

using namespace std;

//const double PI = 3.141592653589793238460;
 
typedef std::complex<double> Complex;
typedef std::valarray<Complex> CArray;

__global__ void add(double *re, double *im, int *x, double *odd_real, double *odd_im, double *even_real, double *even_im)
{
    int k = threadIdx.x + blockIdx.x*blockDim.x;//Supposed to count till N/2..
    int N = *x;
    double COS = cos(-2 * 3.141592653589793238460 * k / N);
    double SIN = sin(-2 * 3.141592653589793238460 * k / N);
    //Complex t = std::polar(1.0, -2 * PI * k / N) * odd[k];//(cos0 + isin0)(x + iy) = (...)
    double t_real = odd_real[k]*COS - odd_im[k]*SIN;
    double t_im = odd_im[k]*COS + odd_real[k]*SIN;
    //double sin0 = math.sin(-2 * PI * k / N)
    //use even_real, even_im, odd_real, odd_im
    //x[k    ] = even[k] + t;
    re[k] = even_real[k] + t_real;
    im[k] = even_im[k] + t_im;
    //x[k+N/2] = even[k] - t;
    re[k+N/2] = even_real[k] - t_real;
    im[k+N/2] = even_im[k] - t_im;
}


// Cooley–Tukey FFT (in-place, divide-and-conquer)
// Higher memory requirements and redundancy although more intuitive
void fft(double *re, double *im, int N)//double re[], double im[]
{
    if (N <= 1) return;
 
    // divide
    int size = N/2*sizeof(double);

    double *even_real;
    even_real = (double*)malloc(size);
    double *even_im;
    even_im = (double*)malloc(size);
    double *odd_real;
    odd_real = (double*)malloc(size);
    double *odd_im;
    odd_im = (double*)malloc(size);
    for(int i = 0; i < N; i++){
        if(i%2==0){
            even_real[i/2] = re[i];
            even_im[i/2] = im[i];
        }
        else{
            odd_real[(i-1)/2] = re[i];
            odd_im[(i-1)/2] = im[i];
        }
    }
 
    // conquer
    fft(even_real, even_im, N/2);//even_real, even_im
    fft(odd_real, odd_im, N/2);//odd_real, odd_im
 
    // combine

    // taking 8 blocks and N/8 threads

    int num_threads = N/16; //4096/4 = 1024 = max num of threads
    int num_blocks = 8;
    if(N < 16)
    {
        num_threads = N;
        num_blocks = 1;
    }

    double *dev_re, *dev_im, *dev_odd_real, *dev_odd_im, *dev_even_real, *dev_even_im;
    hipMalloc( (void**)&dev_re, 2*size );
    hipMalloc( (void**)&dev_im, 2*size );
    hipMalloc( (void**)&dev_odd_real, size );
    hipMalloc( (void**)&dev_odd_im, size );
    hipMalloc( (void**)&dev_even_real, size );
    hipMalloc( (void**)&dev_even_im, size );
    int *dev_N;
    hipMalloc( (void**)&dev_N, sizeof(int) );
    
    hipMemcpy( dev_re, re, 2*size, hipMemcpyHostToDevice );
    hipMemcpy( dev_im, im, 2*size, hipMemcpyHostToDevice );
    hipMemcpy( dev_odd_real, odd_real, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_odd_im, odd_im, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_even_im, even_im, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_even_real, even_real, size, hipMemcpyHostToDevice );
    hipMemcpy( dev_N, &N, sizeof(int), hipMemcpyHostToDevice );


    add<<< num_blocks , num_threads >>>(dev_re, dev_im, dev_N, dev_odd_real, dev_odd_im, dev_even_real, dev_even_im);

    hipMemcpy( im, dev_im, 2*size, hipMemcpyDeviceToHost );
    hipMemcpy( re, dev_re, 2*size, hipMemcpyDeviceToHost );

    free(even_real);
    free(even_im);
    free(odd_real);
    free(odd_im);

    hipFree(dev_im);
    hipFree(dev_re);
    hipFree(dev_odd_real);
    hipFree(dev_even_real);
    hipFree(dev_odd_im);
    hipFree(dev_even_im);
    hipFree(dev_N);
}
 


double magnitude(complex<double> p, int N) 
{ 
	return 2*sqrt(pow(p.real(),2) + pow(p.imag(),2))/N;
}




/* truncate very small numbers to 0 */
double approx_zero(double d) 
{ 
	if (abs(d) < 0.0000000000001)
		return 0;
	else 
		return d;
}





int main(){
    
    const int sr=16384;
    //enter sampling rate 

    double ar[sr*26];
    //26 second song, sr data points per second, or sr hz sampling frequency
    ifstream inFile;
    inFile.open("data.txt");
    int i = 0;
    double value;
    while(inFile >> value){
        ar[i] = value;
        i++;
    }

    //vector containing max amplitude with frequency
    vector<pair<int,double>> dft;

    ofstream outdata;
	outdata.open("data2.txt");


    for(i = 0; i < 26; i++){

        Complex test[sr];
        for(int j=0;j<sr;j++){
        test[j]=ar[i*sr+j];
        }

        CArray s(test, sr);// have to change.
        
        //convert for fft
        double *re;
        re = (double*)malloc(sr*sizeof(double));
        double *im;
        im = (double*)malloc(sr*sizeof(double));
        for(int j = 0; j < sr; j++){
            re[j] = real(s[j]);
            im[j] = imag(s[j]);
        }
        //forward fft
        fft(re, im, sr);

        //convert back to CArray
        for(int j = 0; j < sr; j++){
            Complex temp(re[j], im[j]);
            s[j] = temp;
        }

	    int idx=0;
	    outdata.precision(4);
        outdata<<"iteration: "<< i+1 <<endl;
	    for (idx=0; idx < sr; idx++)
		  outdata << idx << " ";
	    outdata << endl;
	    for (idx=0; idx < sr; idx++)
	    	outdata << (magnitude(s[idx],sr)) << " ";
	    outdata << endl <<endl;
        
        int xk = 0;
        int abs=0.0000000000001;
        for(int j = 1; j <sr/2; j++){
            if(magnitude(s[xk],sr)-(magnitude(s[j],sr))<abs)
                xk=j;
                //Add real part
                //xk += ar[i*1000 + j] * cos(2*M_PI*k*j/1054);
            }
        dft.push_back(make_pair(xk,magnitude(s[xk],sr)));
        //Find 1 highest frequency in each second..          
        
    }
    outdata.close();
    for(auto a:dft)
    {
     	cout<<a.first<<" "<<a.second<<"\n";
    }
    return 0;
}
